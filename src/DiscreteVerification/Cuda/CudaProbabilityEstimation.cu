#include "hip/hip_runtime.h"
#include "DiscreteVerification/Alloc/CudaPetriNetAllocator.cuh"
#include "DiscreteVerification/Cuda/CudaAST.cuh"
#include "DiscreteVerification/Cuda/CudaQueryVisitor.cuh"
#include "DiscreteVerification/Cuda/CudaRunResult.cuh"
#include "DiscreteVerification/Cuda/CudaSMCQueryConverter.cuh"
#include "DiscreteVerification/Cuda/CudaTAPNConverter.cuh"
#include "DiscreteVerification/VerificationTypes/AtlerProbabilityEstimation.hpp"

#include <hip/hip_runtime.h>

namespace VerifyTAPN::DiscreteVerification {
__global__ void runSimulationKernel(Cuda::CudaTimedArcPetriNet *ctapn, Cuda::CudaRealMarking *initialMarking,
                                    Cuda::AST::CudaSMCQuery *query, Cuda::CudaRunResult *runner, int *timeBound,
                                    int *stepBound, int *successCount, int *runsNeeded, hiprandState *states) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int runNeed = *runsNeeded;
  if (tid >= runNeed) return;

  hiprand_init(clock64(), tid, 0, &states[tid]);
  if (tid % 1000 == 0) {
    printf("Thread %d initialized\n", tid);
  }

  int tBound = *timeBound;
  int sBound = *stepBound;

  Cuda::CudaTimedArcPetriNet tapn = *ctapn;

  // TODO prepare per thread
  // runner.prepare(initialMarking);
  Cuda::CudaRealMarking *newMarking = runner->parent;

  while (!runner->maximal && !(runner->totalTime >= tBound || runner->totalSteps >= sBound)) {

    Cuda::CudaRealMarking *child = newMarking->clone();
    Cuda::CudaQueryVisitor checker(*child, tapn);
    Cuda::AST::BoolResult result;

    query->accept(checker, result);

    if (result.value) {
      atomicAdd(successCount, 1);
      break;
    }
    newMarking = runner->next(tid);
  }
}

__global__ void testAllocationKernel(VerifyTAPN::Cuda::CudaTimedArcPetriNet* pn){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(tid == 0){
  printf("Petri net maxConstant: %s\n", pn->maxConstant);
  printf("Petri net placesLength: %d\n", pn->placesLength);
  for(int i = 0; i < pn->placesLength; i++){
    printf("Place id: %d\n", pn->places[i]->id);
    printf("Place name: %s\n", pn->places[i]->name);
    printf("Place invariant: %s\n", pn->places[i]->timeInvariant);
    printf("Place type: %s\n", pn->places[i]->type);
    printf("Place containsInhibitorArcs: %s\n", pn->places[i]->containsInhibitorArcs);
    printf("Place inputArcsLength: %d\n", pn->places[i]->inputArcsLength);
    for(int x = 0; x < pn->places[i]->inputArcsLength; x++){
      printf("Place inputArcs[%d]: %s\n", x, pn->places[i]->inputArcs[x]->weight);
    }
  }
  printf("Kernel executed\n");
  }
};

bool AtlerProbabilityEstimation::runCuda() {
  std::cout << "Converting TAPN and marking..." << std::endl;
  auto result = VerifyTAPN::Cuda::CudaTAPNConverter::convert(tapn, initialMarking);
  VerifyTAPN::Cuda::CudaTimedArcPetriNet ctapn = result->first;
  VerifyTAPN::Cuda::CudaRealMarking ciMarking = result->second;

  std::cout << "Converting Query..." << std::endl;
  SMCQuery *currentSMCQuery = static_cast<SMCQuery *>(query);
  VerifyTAPN::Cuda::AST::CudaSMCQuery *cudaSMCQuery = VerifyTAPN::Cuda::CudaSMCQueryConverter::convert(currentSMCQuery);

  // std::cout << "Converting Options..." << std::endl;
  // VerifyTAPN::Cuda::CudaVerificationOptions cudaOptions = Cuda::CudaOptionsConverter::convert(options);

  // TODO: Convert the PlaceVisitor to a simple representation
  // NOTE: Also find a way to simplify the representation of the PlaceVisitor

  std::cout << "Creating run generator..." << std::endl;

  const unsigned int threadsPerBlock = 256;

  // Calculate the number of blocks needed
  unsigned int blocks = (this->runsNeeded + threadsPerBlock - 1) / threadsPerBlock;

  std::cout << "Runs needed..." << this->runsNeeded << std::endl;
  std::cout << "Threads per block..." << threadsPerBlock << std::endl;
  std::cout << "Blocks..." << blocks << std::endl;

  // Allocate the petry net

  VerifyTAPN::Alloc::CudaPetriNetAllocator pnAllocator;

  VerifyTAPN::Cuda::CudaTimedArcPetriNet* pn = pnAllocator.cuda_allocator(&ctapn);

  testAllocationKernel<<<blocks, threadsPerBlock>>>(pn);
  // Allocate the initial marking

  //Allocate the query

  hipError_t allocStatus = hipGetLastError();
  if (allocStatus != hipSuccess) {
    std::cerr << "hipMalloc failed: " << hipGetErrorString(allocStatus) << std::endl;
  } else {
    std::cout << "Device memory for hiprand allocated successfully." << std::endl;
  }

  auto runres = new VerifyTAPN::Cuda::CudaRunResult(ctapn);

  std::cout << "Run prepare" << std::endl;

  // VerifyTAPN::DiscreteVerification::runSimulationKernel<<<blocks, threads>>>(
  //     stapn, ciMarking, cudaSMCQuery, runres, smcSettings.timeBound, smcSettings.stepBound, 0, runsNeeded);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
    return false;
  }

  err = hipDeviceSynchronize();

  if (err != hipSuccess) {
    std::cerr << "CUDA device synchronization failed: " << hipGetErrorString(err) << std::endl;
    return false;
  }

  std::cout << "Kernel execution completed successfully." << std::endl;

  return false;
}
} // namespace VerifyTAPN::DiscreteVerification