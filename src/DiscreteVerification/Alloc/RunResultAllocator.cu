
#include "DiscreteVerification/Cuda/CudaRunResult.cuh"
#include "DiscreteVerification/Alloc/RunResultAllocator.cuh"

#include <hip/hip_runtime.h>

namespace VerifyTAPN::Alloc {

using namespace Cuda;

__host__ CudaRunResult* allocate(CudaRunResult *runResultHost, int blocks, int threadsPerBlock) {
  int numThreads = blocks * threadsPerBlock;

  // Allocate device memory for rngStates
  hipMalloc(&(runResultHost->rngStates), numThreads * sizeof(hiprandState_t));

  // Allocate device memory for CudaRunResult
  CudaRunResult *runResultDevice;
  hipMalloc(&runResultDevice, sizeof(CudaRunResult));

  // Copy CudaRunResult from host to device
  hipMemcpy(runResultDevice, runResultHost, sizeof(CudaRunResult), hipMemcpyHostToDevice);

  return runResultDevice;
}

__host__ void allocatePointerMembers(CudaRunResult *runResultHost) {
  // Allocate single oduble for dates_sampled which is an pointer to dynamic array of doubles 
  /* Look into this later */
  hipMalloc(&runResultHost->dates_sampled, sizeof(double));

  // Allocate marking for parent
  

  // Allocate marking for origin




  // Allocate array of places for markings
}

} // namespace VerifyTAPN::Alloc